#include "hip/hip_runtime.h"
#include "libejovo.hpp"

#include <complex>

// Unified matrix type
// unified matrix of floats
class UniMatrix : public ejovo::Grid2D<float> {


public:

    UniMatrix(int m, int n);
    ~UniMatrix();

    // Override pure virtual functions
    float& operator[](int i) override;
    const float& operator[](int i) const override;
    std::size_t ncol() const override;
    std::size_t nrow() const override;

    ejovo::Matrix<float> to_matrix() const override;

    float *data;
    std::size_t m;
    std::size_t n;


};

float& UniMatrix::operator[](int i) {
    return data[i];
}

const float& UniMatrix::operator[](int i) const {
    return data[i];
}

std::size_t UniMatrix::ncol() const {
    return n;
}

std::size_t UniMatrix::nrow() const {
    return m;
}

ejovo::Matrix<float> UniMatrix::to_matrix() const {
    ejovo::Matrix<float> out (m, n);
    out.loop_i([&] (int i) {
        out(i) = this->operator()(i);
    });
    return out;
}


UniMatrix::UniMatrix(int __m, int __n)
    : m{__m}
    , n{__n}
    , data{nullptr}
{
    float *x;
    hipMallocManaged(&x, (__m * __n) * sizeof(float));
    data = x;
}

// free the associated memory
UniMatrix::~UniMatrix() {
    if (!data) hipFree(data);
}


__global__ void touch_data(int n, float *A) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        A[i] = 10;
    }

}

__global__ void touch_matrix(UniMatrix& A) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < A.size(); i += stride) {
        A[i] = 10;
    }


}


// Create a unified zeros matrix

// ejovo::Matrix<float> unified_matrix(int m, int n) {

//     ejovo::Matrix<float> m;

//     float *unified_data;
//     hipMallocManaged(&unified_data, (m * n) * sizeof(float));

//     // m.data = std::unique_ptr




// }


int main() {

    using namespace ejovo;

    std::cout << "Hello world\n";

    ejovo::Matrix<double> m = ejovo::Matrix<double>::zeros(10, 3);
    m.print();

    const auto& [th, ph] = meshgrid<double>(linspace<double>(0.0, trig::two_pi, 10), linspace<double>(0.0, trig::pi, 10));


    auto Y01 = [&] (double th, double ph) {
        return 0.5 * sqrt(3.0 / trig::two_pi) * cos(th);
    };

    std::cout << "Azimuthal angle theta:\n";

    th.print();

    std::cout << "Polar angle phi: \n";

    ph.print();

    auto sph_harm = map2<double>(th, ph, Y01);

    std::cout << "Y01: \n";

    sph_harm.print();

    // contour(th, ph, sph_harm);

    // create memory
    // double *

    // Let me go ahead an compute some spherical harmonic functions...

    auto um = UniMatrix(10, 10);
    um.print();

    touch_data<<<1, 1>>>(100, um.data);

    um.print();

    um.fill(10); // This proves that the host can access the memory.

    um.print();



}